#include "hip/hip_runtime.h"

#include "gtest/gtest.h"

#include "../libpsc/cuda/cuda_mparticles.cuh"
#include "../libpsc/cuda/cuda_mparticles_sort.cuh"

#include "../libpsc/cuda/collision_cuda_impl.hxx"
#include "../libpsc/cuda/cuda_collision.cuh"
#include "psc_fields_single.h"
#include "psc_particles_single.h"
#include "testing.hxx"

using dim = dim_yz;
using Mparticles = MparticlesCuda<BS144>;

// ======================================================================
// RngStateCuda

TEST(RngStateCuda, ctor_dtor)
{
  RngStateCuda rng_state;
  RngStateCuda rng_state2(128);
}

TEST(RngStateCuda, resize)
{
  RngStateCuda rng_state;
  rng_state.resize(256);
  EXPECT_EQ(rng_state.size(), 256);
}

__global__ static void kernel_random(RngStateCuda::Device rng_state,
                                     thrust::device_ptr<float> x)
{
  int n = threadIdx.x + blockDim.x * blockIdx.x;

  auto rng = rng_state[n];
  x[n] = rng.uniform();
  rng_state[n] = rng;
}

TEST(RngStateCuda, access)
{
  dim3 dim_grid(2);
  RngStateCuda rng_state(dim_grid.x * THREADS_PER_BLOCK);

  ASSERT_EQ(THREADS_PER_BLOCK, 128);
  psc::device_vector<float> x(dim_grid.x * THREADS_PER_BLOCK);
  kernel_random<<<dim_grid, THREADS_PER_BLOCK>>>(rng_state, x.data());

  float sum = thrust::reduce(x.begin(), x.end(), 0.f, thrust::plus<float>());
  float avg = sum / x.size();
  EXPECT_NEAR(avg, .5, .05);

  // repeat to make sure numbers don't repeat
  kernel_random<<<dim_grid, THREADS_PER_BLOCK>>>(rng_state, x.data());

  float sum2 = thrust::reduce(x.begin(), x.end(), 0.f, thrust::plus<float>());
  float avg2 = sum2 / x.size();
  EXPECT_NEAR(avg2, .5, .05);
  EXPECT_NE(avg, avg2);
}

// ======================================================================
// make_psc
//
// FIXME, duplicated in various testing environments

template <typename dim>
static Grid_t& make_psc(const Grid_t::Kinds& kinds)
{
  Int3 gdims = {16, 16, 16};
  Int3 ibn = {2, 2, 2};
  Int3 np = {2, 2, 2};
  Vec3<double> length = {160., 160., 160.};
  if (dim::InvarX::value) {
    gdims[0] = 1;
    ibn[0] = 0;
    np[0] = 1;
  }
  if (dim::InvarY::value) {
    gdims[1] = 1;
    ibn[1] = 0;
    np[1] = 1;
  }
  if (dim::InvarZ::value) {
    gdims[2] = 1;
    ibn[2] = 0;
    np[2] = 1;
  }

  auto grid_domain = Grid_t::Domain{gdims, length, {}, np};
  auto grid_bc =
    psc::grid::BC{{BND_FLD_PERIODIC, BND_FLD_PERIODIC, BND_FLD_PERIODIC},
                  {BND_FLD_PERIODIC, BND_FLD_PERIODIC, BND_FLD_PERIODIC},
                  {BND_PRT_PERIODIC, BND_PRT_PERIODIC, BND_PRT_PERIODIC},
                  {BND_PRT_PERIODIC, BND_PRT_PERIODIC, BND_PRT_PERIODIC}};

  auto norm_params = Grid_t::NormalizationParams::dimensionless();
  norm_params.nicell = 200;
  auto coeff = Grid_t::Normalization{norm_params};
  return *new Grid_t{grid_domain, grid_bc, kinds, coeff, 1.};
}

static Mparticles make_mparticles(const Grid_t& grid)
{
  Mparticles mprts{grid};
  auto inj = mprts.injector();
  auto injector = inj[0];
  injector({{5., 5., 5.}, {1., 0., 0.}, 1., 0});
  injector({{5., 5., 5.}, {0., 0., 0.}, 1., 0});
  injector({{5., 15., 15.}, {.6, 0., 0.}, 1., 0});
  injector({{5., 15., 15.}, {.7, 0., 0.}, 1., 0});
  injector({{5., 15., 15.}, {.8, 0., 0.}, 1., 0});
  injector({{5., 15., 5.}, {.1, 0., 0.}, 1., 0});
  injector({{5., 15., 5.}, {.2, 0., 0.}, 1., 0});
  injector({{5., 15., 5.}, {.3, 0., 0.}, 1., 0});
  injector({{5., 15., 5.}, {.4, 0., 0.}, 1., 0});
  MHERE;
  return mprts;
}

TEST(cuda_mparticles_sort, sort)
{
  auto kinds = Grid_t::Kinds{Grid_t::Kind(1., 1., "test_species")};
  const auto& grid = make_psc<dim>(kinds);
  std::cout << "n_patches " << grid.n_patches() << "\n";

  // init particles
  MHERE;
  auto mprts = make_mparticles(grid);
  MHERE;

  auto& cmprts = *mprts.cmprts();
  auto sort = cuda_mparticles_sort(cmprts.n_cells());

  sort.find_indices_ids(cmprts);
  EXPECT_EQ(sort.d_idx, (std::vector<int>{0, 0, 9, 9, 9, 1, 1, 1, 1}));
  EXPECT_EQ(sort.d_id, (std::vector<int>{0, 1, 2, 3, 4, 5, 6, 7, 8}));

  sort.stable_sort_cidx();
  EXPECT_EQ(sort.d_idx, (std::vector<int>{0, 0, 1, 1, 1, 1, 9, 9, 9}));
  EXPECT_EQ(sort.d_id, (std::vector<int>{0, 1, 5, 6, 7, 8, 2, 3, 4}));

  sort.find_offsets();
  std::vector<int> off(cmprts.n_cells() + 1);
  off[0] = 0;
  off[1] = 2;
  for (int i = 2; i < 10; i++) {
    off[i] = 6;
  }
  for (int i = 10; i <= 256; i++) {
    off[i] = 9;
  }
  EXPECT_EQ(sort.d_off, off);
}

TEST(cuda_mparticles_randomize_sort, sort)
{
  auto kinds = Grid_t::Kinds{Grid_t::Kind(1., 1., "test_species")};
  const auto& grid = make_psc<dim>(kinds);

  // init particles
  auto mprts = make_mparticles(grid);

  auto& cmprts = *mprts.cmprts();
  cuda_mparticles_randomize_sort sort;

  sort.find_indices_ids(cmprts);
  EXPECT_EQ(sort.d_id, (std::vector<int>{0, 1, 2, 3, 4, 5, 6, 7, 8}));

  sort.sort();
  // EXPECT_EQ(sort.d_id, (std::vector<int>{0, 1, 5, 8, 7, 6, 2, 3, 4}));
  // EXPECT_EQ(sort.d_id, (std::vector<int>{1, 0, 8, 7, 5, 6, 4, 2, 3}));
  EXPECT_EQ(sort.d_id, (std::vector<int>{0, 1, 7, 5, 8, 6, 2, 4, 3}));

  float last = sort.d_random_idx[0];
  for (int i = 1; i < cmprts.size(); i++) {
    EXPECT_GE(sort.d_random_idx[i], last);
    last = sort.d_random_idx[i];
  }
  // for (int i = 0; i < cmprts.size(); i++) {
  //   mprintf("i %d r_idx %g id %d\n", i, (float)sort.d_random_idx[i],
  //           (int)sort.d_id[i]);
  // }

  sort.find_offsets();
  std::vector<int> off(cmprts.n_cells() + 1);
  off[0] = 0;
  off[1] = 2;
  for (int i = 2; i < 10; i++) {
    off[i] = 6;
  }
  for (int i = 10; i <= 256; i++) {
    off[i] = 9;
  }
  EXPECT_EQ(sort.d_off, off);

#if 1
  // do over, get different permutation
  sort.find_indices_ids(cmprts);
  sort.sort();
  // for (int i = 0; i < cmprts.size(); i++) {
  //   mprintf("i %d r_idx %g id %d\n", i, (float)sort.d_random_idx[i],
  //           (int)sort.d_id[i]);
  // }
  EXPECT_NE(sort.d_id, (std::vector<int>{0, 1, 7, 5, 8, 6, 2, 4, 3}));
#endif
}

TEST(CollisionTest, Test2)
{
  using Collision = CollisionCuda<MparticlesCuda<BS144>, RngStateFake>;
  const typename Mparticles::real_t eps = 1e-5;

  auto kinds = Grid_t::Kinds{Grid_t::Kind(1., 1., "test_species")};
  const auto& grid = make_psc<dim>(kinds);

  // init particles
  auto mprts = make_mparticles(grid);

  auto collision = Collision(grid, 1, 1.);

  auto& cmprts = *mprts.cmprts();
  auto sort_by_cell = cuda_mparticles_sort(cmprts.n_cells());
  sort_by_cell.find_indices_ids(cmprts);
  EXPECT_EQ(sort_by_cell.d_idx, (std::vector<int>{0, 0, 9, 9, 9, 1, 1, 1, 1}));
  EXPECT_EQ(sort_by_cell.d_id, (std::vector<int>{0, 1, 2, 3, 4, 5, 6, 7, 8}));
  sort_by_cell.stable_sort_cidx();
  EXPECT_EQ(sort_by_cell.d_idx, (std::vector<int>{0, 0, 1, 1, 1, 1, 9, 9, 9}));
  EXPECT_EQ(sort_by_cell.d_id, (std::vector<int>{0, 1, 5, 6, 7, 8, 2, 3, 4}));
  sort_by_cell.find_offsets();
  // for (int i = 0; i < cmprts.size(); i++) {
  //   mprintf("i %d idx %d id %d\n", i, (int)sort_by_cell.d_idx[i],
  //   (int)sort_by_cell.d_id[i]);

  collision(mprts);

  auto accessor = mprts.accessor();
  auto it = accessor[0].begin();
  auto prtf0 = *it++;
  auto prtf1 = *it++;
  EXPECT_NEAR(prtf0.u()[0] + prtf1.u()[0], 1., eps);
  EXPECT_NEAR(prtf0.u()[1] + prtf1.u()[1], 0., eps);
  EXPECT_NEAR(prtf0.u()[2] + prtf1.u()[2], 0., eps);

#if 0
  // depends on random numbers, but for RngFake, we know
  EXPECT_NEAR(prtf0.u()[0],  0.96226911, eps);
  EXPECT_NEAR(prtf0.u()[1],  0.        , eps);
  EXPECT_NEAR(prtf0.u()[2], -0.17342988, eps);
  EXPECT_NEAR(prtf1.u()[0],  0.03773088, eps);
  EXPECT_NEAR(prtf1.u()[1], -0.        , eps);
  EXPECT_NEAR(prtf1.u()[2],  0.17342988, eps);
#endif
}

// ======================================================================
// main

int main(int argc, char** argv)
{
  MPI_Init(&argc, &argv);
  ::testing::InitGoogleTest(&argc, argv);
  cuda_base_init();
  int rc = RUN_ALL_TESTS();
  MPI_Finalize();
  return rc;
}
