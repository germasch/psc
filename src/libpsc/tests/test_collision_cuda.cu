#include "hip/hip_runtime.h"

#include "gtest/gtest.h"

#include "../libpsc/cuda/cuda_mparticles.cuh"
#include "../libpsc/cuda/cuda_mparticles_sort.cuh"

#include "../libpsc/cuda/collision_cuda_impl.hxx"
#include "../libpsc/cuda/cuda_collision.cuh"
#include "psc_fields_single.h"
#include "psc_particles_single.h"
#include "testing.hxx"

using dim = dim_yz;
using BS = BS144;
using Mparticles = MparticlesCuda<BS>;

// ======================================================================
// RngStateCuda

TEST(RngStateCuda, ctor_dtor)
{
  RngStateCuda rng_state;
  RngStateCuda rng_state2(128);
}

TEST(RngStateCuda, resize)
{
  RngStateCuda rng_state;
  rng_state.resize(256);
  EXPECT_EQ(rng_state.size(), 256);
}

__global__ static void kernel_random(RngStateCuda::Device rng_state,
                                     thrust::device_ptr<float> x)
{
  int n = threadIdx.x + blockDim.x * blockIdx.x;

  auto rng = rng_state[n];
  x[n] = rng.uniform();
  rng_state[n] = rng;
}

TEST(RngStateCuda, access)
{
  dim3 dim_grid(2);
  RngStateCuda rng_state(dim_grid.x * THREADS_PER_BLOCK);

  ASSERT_EQ(THREADS_PER_BLOCK, 128);
  psc::device_vector<float> x(dim_grid.x * THREADS_PER_BLOCK);
  kernel_random<<<dim_grid, THREADS_PER_BLOCK>>>(rng_state, x.data());

  float sum = thrust::reduce(x.begin(), x.end(), 0.f, thrust::plus<float>());
  float avg = sum / x.size();
  EXPECT_NEAR(avg, .5, .05);

  // repeat to make sure numbers don't repeat
  kernel_random<<<dim_grid, THREADS_PER_BLOCK>>>(rng_state, x.data());

  float sum2 = thrust::reduce(x.begin(), x.end(), 0.f, thrust::plus<float>());
  float avg2 = sum2 / x.size();
  EXPECT_NEAR(avg2, .5, .05);
  EXPECT_NE(avg, avg2);
}

// ======================================================================
// make_psc
//
// FIXME, duplicated in various testing environments

template <typename dim>
static Grid_t& make_psc(const Grid_t::Kinds& kinds)
{
  Int3 gdims = {16, 16, 16};
  Int3 ibn = {2, 2, 2};
  Int3 np = {2, 2, 2};
  Vec3<double> length = {160., 160., 160.};
  if (dim::InvarX::value) {
    gdims[0] = 1;
    ibn[0] = 0;
    np[0] = 1;
  }
  if (dim::InvarY::value) {
    gdims[1] = 1;
    ibn[1] = 0;
    np[1] = 1;
  }
  if (dim::InvarZ::value) {
    gdims[2] = 1;
    ibn[2] = 0;
    np[2] = 1;
  }

  auto grid_domain = Grid_t::Domain{gdims, length, {}, np};
  auto grid_bc =
    psc::grid::BC{{BND_FLD_PERIODIC, BND_FLD_PERIODIC, BND_FLD_PERIODIC},
                  {BND_FLD_PERIODIC, BND_FLD_PERIODIC, BND_FLD_PERIODIC},
                  {BND_PRT_PERIODIC, BND_PRT_PERIODIC, BND_PRT_PERIODIC},
                  {BND_PRT_PERIODIC, BND_PRT_PERIODIC, BND_PRT_PERIODIC}};

  auto norm_params = Grid_t::NormalizationParams::dimensionless();
  norm_params.nicell = 200;
  auto coeff = Grid_t::Normalization{norm_params};
  return *new Grid_t{grid_domain, grid_bc, kinds, coeff, 1.};
}

static Mparticles make_mparticles(const Grid_t& grid)
{
  //                                   y
  //     10  20  30  40  50  60  70  80    90 100 110 120 130 140 150 160
  // +---+---+---+---+---+---+---+---+ +---+---+---+---+---+---+---+---+ 0
  // | AB|FGHI   |   |   |   |   |   | |   |   |   |   |   |   |   |   |
  // +---+---+---+---+---+---+---+---+ +---+---+---+---+---+---+---+---+ 10
  // |   |CDE|   |   |   |   |   |   | |   |   |   |   |   |   |   |   |
  // +---+---+---+---+---+---+---+---+ +---+---+---+---+---+---+---+---+ 20  z
  // |   |   |   |   |   |   |   |   | |   |   | IJ|NO |   |   |   |   |
  // +---+---+---+---+---+---+---+---+ +---+---+---+---+---+---+---+---+ 30
  // |   |   |   |   |   |   |   |   | |   |   |   |LM |   |   |   |   |
  // +---+---+---+---+---+---+---+---+ +---+---+---+---+---+---+---+---+ 40

  Mparticles mprts{grid};
  auto inj = mprts.injector();
  {
    auto injector = inj[0];
    injector({{5., 5., 5.}, {0., 0., 0.}, 1., 0});    // A
    injector({{5., 5., 5.}, {0.01, 0., 0.}, 1., 0});  // B
    injector({{5., 15., 15.}, {.02, 0., 0.}, 1., 0}); // C
    injector({{5., 15., 15.}, {.03, 0., 0.}, 1., 0}); // D
    injector({{5., 15., 15.}, {.04, 0., 0.}, 1., 0}); // E
    injector({{5., 15., 5.}, {.05, 0., 0.}, 1., 0});  // F
    injector({{5., 15., 5.}, {.06, 0., 0.}, 1., 0});  // G
    injector({{5., 15., 5.}, {.07, 0., 0.}, 1., 0});  // H
    injector({{5., 15., 5.}, {.08, 0., 0.}, 1., 0});  // I
  }
  {
    auto injector = inj[1];
    injector({{5., 105., 25.}, {.09, 0., 0.}, 1., 0}); // J
    injector({{5., 105., 25.}, {.10, 0., 0.}, 1., 0}); // K
    injector({{5., 115., 35.}, {.11, 0., 0.}, 1., 0}); // L
    injector({{5., 115., 35.}, {.12, 0., 0.}, 1., 0}); // M
    injector({{5., 115., 25.}, {.13, 0., 0.}, 1., 0}); // N
    injector({{5., 115., 25.}, {.14, 0., 0.}, 1., 0}); // O
  }
  return mprts;
}

TEST(cuda_mparticles_sort, sort)
{
  auto kinds = Grid_t::Kinds{Grid_t::Kind(1., 1., "test_species")};
  const auto& grid = make_psc<dim>(kinds);

  // init particles
  auto mprts = make_mparticles(grid);

  auto& cmprts = *mprts.cmprts();
  std::cout << "off sz " << cmprts.by_block_.d_off.size() << "\n";
  std::cout << "off ";
  std::copy(cmprts.by_block_.d_off.begin(), cmprts.by_block_.d_off.end(),
            std::ostream_iterator<double>(std::cout, " "));
  std::cout << "\n";

  auto sort = cuda_mparticles_sort(cmprts.n_cells());

  sort.find_indices_ids<BS, dim>(cmprts);
  EXPECT_EQ(sort.d_idx, (std::vector<int>{0, 0, 9, 9, 9, 1, 1, 1, 1, 82, 82, 91,
                                          91, 83, 83}));
  EXPECT_EQ(sort.d_id, (std::vector<int>{0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11,
                                         12, 13, 14}));

  sort.stable_sort_cidx();
  EXPECT_EQ(sort.d_idx, (std::vector<int>{0, 0, 1, 1, 1, 1, 9, 9, 9, 82, 82, 83,
                                          83, 91, 91}));
  EXPECT_EQ(sort.d_id, (std::vector<int>{0, 1, 5, 6, 7, 8, 2, 3, 4, 9, 10, 13,
                                         14, 11, 12}));

  sort.find_offsets();
  std::vector<int> off(cmprts.n_cells() + 1);
  off[0] = 0;
  off[1] = 2;
  for (int i = 2; i < 10; i++) {
    off[i] = 6;
  }
  for (int i = 10; i < 83; i++) {
    off[i] = 9;
  }
  for (int i = 83; i < 84; i++) {
    off[i] = 11;
  }
  for (int i = 84; i < 92; i++) {
    off[i] = 13;
  }
  for (int i = 92; i <= 256; i++) {
    off[i] = 15;
  }
  std::copy(sort.d_off.begin(), sort.d_off.end(),
            std::ostream_iterator<int>(std::cout, " "));
  std::cout << "\n";
  EXPECT_EQ(sort.d_off, off);
}

TEST(cuda_mparticles_randomize_sort, sort)
{
  auto kinds = Grid_t::Kinds{Grid_t::Kind(1., 1., "test_species")};
  const auto& grid = make_psc<dim>(kinds);

  // init particles
  auto mprts = make_mparticles(grid);

  auto& cmprts = *mprts.cmprts();
  cuda_mparticles_randomize_sort sort;

  sort.find_indices_ids(cmprts);
  EXPECT_EQ(sort.d_id, (std::vector<int>{0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11,
                                         12, 13, 14}));

  sort.sort();
  EXPECT_EQ(sort.d_id, (std::vector<int>{0, 1, 7, 5, 8, 6, 2, 4, 3, 9, 10, 13,
                                         14, 11, 12}));

  auto last = sort.d_random_idx[0];
  for (int i = 1; i < cmprts.size(); i++) {
    EXPECT_GE(sort.d_random_idx[i], last);
    last = sort.d_random_idx[i];
  }
  // std::cout << "rnd idx ";
  // std::copy(sort.d_random_idx.begin(), sort.d_random_idx.end(),
  //           std::ostream_iterator<double>(std::cout, " "));
  // std::cout << "\n";

  sort.find_offsets();
  std::vector<int> off(cmprts.n_cells() + 1);
  off[0] = 0;
  off[1] = 2;
  for (int i = 2; i < 10; i++) {
    off[i] = 6;
  }
  for (int i = 10; i < 83; i++) {
    off[i] = 9;
  }
  for (int i = 83; i < 84; i++) {
    off[i] = 11;
  }
  for (int i = 84; i < 92; i++) {
    off[i] = 13;
  }
  for (int i = 92; i <= 256; i++) {
    off[i] = 15;
  }
  std::copy(sort.d_off.begin(), sort.d_off.end(),
            std::ostream_iterator<int>(std::cout, " "));
  std::cout << "\n";
  EXPECT_EQ(sort.d_off, off);

#if 1
  // do over, get different permutation
  sort.find_indices_ids(cmprts);
  sort.sort();
  // for (int i = 0; i < cmprts.size(); i++) {
  //   mprintf("i %d r_idx %g id %d\n", i, (float)sort.d_random_idx[i],
  //           (int)sort.d_id[i]);
  // }
  EXPECT_NE(sort.d_id, (std::vector<int>{0, 1, 7, 5, 8, 6, 2, 4, 3}));
#endif
}

TEST(CollisionTest, Test2)
{
  using Collision = CollisionCuda<MparticlesCuda<BS144>, RngStateFake>;
  const typename Mparticles::real_t eps = 1e-5;

  auto kinds = Grid_t::Kinds{Grid_t::Kind(1., 1., "test_species")};
  const auto& grid = make_psc<dim>(kinds);

  // init particles
  auto mprts = make_mparticles(grid);

  auto collision = Collision(grid, 1, 1.);

  auto& cmprts = *mprts.cmprts();
  auto sort_by_cell = cuda_mparticles_sort(cmprts.n_cells());
  sort_by_cell.find_indices_ids<BS, dim>(cmprts);
  EXPECT_EQ(sort_by_cell.d_idx, (std::vector<int>{0, 0, 9, 9, 9, 1, 1, 1, 1, 82,
                                                  82, 91, 91, 83, 83}));
  EXPECT_EQ(sort_by_cell.d_id, (std::vector<int>{0, 1, 2, 3, 4, 5, 6, 7, 8, 9,
                                                 10, 11, 12, 13, 14}));
  sort_by_cell.stable_sort_cidx();
  EXPECT_EQ(sort_by_cell.d_idx, (std::vector<int>{0, 0, 1, 1, 1, 1, 9, 9, 9, 82,
                                                  82, 83, 83, 91, 91}));
  EXPECT_EQ(sort_by_cell.d_id, (std::vector<int>{0, 1, 5, 6, 7, 8, 2, 3, 4, 9,
                                                 10, 13, 14, 11, 12}));
  sort_by_cell.find_offsets();
  // for (int i = 0; i < cmprts.size(); i++) {
  //   mprintf("i %d idx %d id %d\n", i, (int)sort_by_cell.d_idx[i],
  //   (int)sort_by_cell.d_id[i]);

  collision(mprts);

  auto accessor = mprts.accessor();
  auto it = accessor[0].begin();
  auto prtf0 = *it++;
  auto prtf1 = *it++;
  EXPECT_NEAR(prtf0.u()[0] + prtf1.u()[0], 0.01, eps);
  EXPECT_NEAR(prtf0.u()[1] + prtf1.u()[1], 0., eps);
  EXPECT_NEAR(prtf0.u()[2] + prtf1.u()[2], 0., eps);

#if 0
  // depends on random numbers, but for RngFake, we know
  EXPECT_NEAR(prtf0.u()[0],  0.96226911, eps);
  EXPECT_NEAR(prtf0.u()[1],  0.        , eps);
  EXPECT_NEAR(prtf0.u()[2], -0.17342988, eps);
  EXPECT_NEAR(prtf1.u()[0],  0.03773088, eps);
  EXPECT_NEAR(prtf1.u()[1], -0.        , eps);
  EXPECT_NEAR(prtf1.u()[2],  0.17342988, eps);
#endif
}

// ======================================================================
// main

int main(int argc, char** argv)
{
  MPI_Init(&argc, &argv);
  ::testing::InitGoogleTest(&argc, argv);
  cuda_base_init();
  int rc = RUN_ALL_TESTS();
  MPI_Finalize();
  return rc;
}
